#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: ndtrung

#include "QuaternionMath.h"
#include "TwoStepNPTRigidGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNPTRigidGPU.cu
    \brief Defines GPU kernel code for NPT integration on the GPU. Used by TwoStepNPTRigidGPU.
*/

// Flag for invalid particle index, identical to the sentinel value NO_INDEX in RigidData.h
#define INVALID_INDEX 0xffffffff

/*! Maclaurine expansion
    \param x Point to take the expansion

*/
__device__ Scalar nph_maclaurin_series(Scalar x)
    {
    Scalar x2, x4;
    x2 = x * x;
    x4 = x2 * x2;
    return (Scalar(1.0) + Scalar(1.0/6.0) * x2 + Scalar(1.0/120.0) * x4 + Scalar(1.0/5040.0) * x2 * x4 + Scalar(1.0/362880.0) * x4 * x4);
    }

/*! Kernel to zero virial contribution from particles from rigid bodies
    \param d_virial_rigid Virial contribution from particles in rigid bodies
    \param local_num Number of particles in this card
*/
extern "C" __global__ void gpu_nph_rigid_zero_virial_rigid_kernel(Scalar *d_virial_rigid,
                                                                 unsigned int local_num)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // particle's index

    if (idx < local_num)
        {
        d_virial_rigid[idx] = 0;
        }

    }

/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration
    \param rdata_com Body center of mass
    \param d_rigid_group Body group
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total umber of rigid bodies
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata_dilation Volume scaling factor
    \param npt_rdata_dimension System dimensionality
    \param npt_rdata_new box New box sizes
*/

extern "C" __global__ void gpu_nph_rigid_remap_kernel(Scalar4 *rdata_com,
                                                      unsigned int *d_rigid_group,
                                                      unsigned int n_group_bodies,
                                                      unsigned int n_bodies,
                                                      BoxDim box,
                                                      Scalar nph_rdata_dilation,
                                                      unsigned int nph_rdata_dimension,
                                                      Scalar4 *nph_rdata_new_box)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= n_group_bodies)
        return;

    unsigned int idx_body = d_rigid_group[group_idx];

    Scalar3 curL = box.getL();
    Scalar3 L;

    // reset box to new size/shape
    L.x = curL.x * nph_rdata_dilation;
    L.y = curL.y * nph_rdata_dilation;
    if (nph_rdata_dimension == 3)
        L.z = curL.z * nph_rdata_dilation;

    // copy and setL
    BoxDim newBox = box;
    newBox.setL(L);

    Scalar4 com = rdata_com[idx_body];
    Scalar3 f = box.makeFraction(make_scalar3(com.x, com.y, com.z));
    Scalar3 pos = newBox.makeCoordinates(f);

    // write out results
    rdata_com[idx_body] = make_scalar4(pos.x, pos.y, pos.z, 0);

    if (idx_body == 0)
        {
        *(nph_rdata_new_box) = make_scalar4(L.x, L.y, L.z, 0.0f);
        }
    }


#pragma mark RIGID_STEP_ONE_KERNEL
/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration
    \param rdata_com Body center of mass
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_body_image Body image
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param n_group_bodies Number of rigid bodies in my group
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_bodies Total umber of rigid bodies
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param box Box dimensions for periodic boundary condition handling
    \param deltaT Timestep

*/

extern "C" __global__ void gpu_nph_rigid_step_one_body_kernel(Scalar4* rdata_com,
                                                            Scalar4* rdata_vel,
                                                            Scalar4* rdata_angmom,
                                                            Scalar4* rdata_angvel,
                                                            Scalar4* rdata_orientation,
                                                            int3* rdata_body_image,
                                                            Scalar4* rdata_conjqm,
                                                            Scalar *d_rigid_mass,
                                                            Scalar4 *d_rigid_mi,
                                                            Scalar4 *d_rigid_force,
                                                            Scalar4 *d_rigid_torque,
                                                            unsigned int *d_rigid_group,
                                                            unsigned int n_group_bodies,
                                                            unsigned int n_bodies,
                                                            Scalar npt_rdata_epsilon_dot,
                                                            Scalar* npt_rdata_partial_Ksum_t,
                                                            Scalar* npt_rdata_partial_Ksum_r,
                                                            unsigned int npt_rdata_nf_t,
                                                            unsigned int npt_rdata_nf_r,
                                                            unsigned int npt_rdata_dimension,
                                                            BoxDim box,
                                                            Scalar deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= n_group_bodies)
        return;

    // do velocity verlet update
    // v(t+deltaT/2) = v(t) + (1/2)a*deltaT
    // r(t+deltaT) = r(t) + v(t+deltaT/2)*deltaT
    Scalar body_mass;
    Scalar4 moment_inertia, com, vel, orientation, ex_space, ey_space, ez_space, force, torque, conjqm;
    int3 body_image;
    Scalar4 mbody, tbody, fquat;

    Scalar dt_half = Scalar(0.5) * deltaT;
    Scalar onednft, onednfr, tmp, scale_t, scale_r, scale_v, akin_t, akin_r;

    onednft = Scalar(1.0) + (Scalar) (npt_rdata_dimension) / (Scalar) (npt_rdata_nf_t+npt_rdata_nf_r);
    onednfr = (Scalar) (npt_rdata_dimension) / (Scalar) (npt_rdata_nf_t+npt_rdata_nf_r);

    tmp = Scalar(-1.0) * dt_half * onednft * npt_rdata_epsilon_dot;
    scale_t = fast::exp(tmp);
    tmp = Scalar(-1.0) * dt_half * onednfr * npt_rdata_epsilon_dot;
    scale_r = fast::exp(tmp);
    tmp = dt_half * npt_rdata_epsilon_dot;
    scale_v = deltaT * fast::exp(tmp) * nph_maclaurin_series(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    com = rdata_com[idx_body];
    vel = rdata_vel[idx_body];
    orientation = rdata_orientation[idx_body];
    body_image = rdata_body_image[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    conjqm = rdata_conjqm[idx_body];

    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);

    // update velocity
    Scalar dtfm = dt_half / body_mass;

    Scalar4 vel2;
    vel2.x = vel.x + dtfm * force.x;
    vel2.y = vel.y + dtfm * force.y;
    vel2.z = vel.z + dtfm * force.z;
    vel2.x *= scale_t;
    vel2.y *= scale_t;
    vel2.z *= scale_t;
    vel2.w = vel.w;

    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;

    // update position
    Scalar3 pos2;
    pos2.x = com.x + vel2.x * scale_v;
    pos2.y = com.y + vel2.y * scale_v;
    pos2.z = com.z + vel2.z * scale_v;

    // time to fix the periodic boundary conditions
    box.wrap(pos2, body_image);

    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);

    Scalar4 conjqm2;
    conjqm2.x = conjqm.x + deltaT * fquat.x;
    conjqm2.y = conjqm.y + deltaT * fquat.y;
    conjqm2.z = conjqm.z + deltaT * fquat.z;
    conjqm2.w = conjqm.w + deltaT * fquat.w;

    conjqm2.x *= scale_r;
    conjqm2.y *= scale_r;
    conjqm2.z *= scale_r;
    conjqm2.w *= scale_r;

    // use no_squish rotate to update p and q
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(1, conjqm2, orientation, moment_inertia, deltaT);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);

    // update the exyz_space
    // transform p back to angmom
    // update angular velocity
    Scalar4 angmom2;
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);

    angmom2.x *= Scalar(0.5);
    angmom2.y *= Scalar(0.5);
    angmom2.z *= Scalar(0.5);

    Scalar4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);

    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;

    // write out the results (MEM_TRANSFER: ? bytes)
    rdata_com[idx_body] = make_scalar4(pos2.x, pos2.y, pos2.z, com.w);
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_orientation[idx_body] = orientation;
    rdata_body_image[idx_body] = body_image;
    rdata_conjqm[idx_body] = conjqm2;

    npt_rdata_partial_Ksum_t[group_idx] = akin_t;
    npt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step

*/
hipError_t gpu_nph_rigid_step_one(const gpu_rigid_data_arrays& rigid_data,
                                   unsigned int *d_group_members,
                                   unsigned int group_size,
                                   Scalar4 *d_net_force,
                                   const BoxDim& box,
                                   const gpu_npt_rigid_data& npt_rdata,
                                   Scalar deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;

    // setup the grid to run the kernel for rigid bodies
    int block_size = 64;
    int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);
    gpu_nph_rigid_step_one_body_kernel<<< body_grid, body_threads  >>>(rigid_data.com,
                                                                       rigid_data.vel,
                                                                       rigid_data.angmom,
                                                                       rigid_data.angvel,
                                                                       rigid_data.orientation,
                                                                       rigid_data.body_image,
                                                                       rigid_data.conjqm,
                                                                       rigid_data.body_mass,
                                                                       rigid_data.moment_inertia,
                                                                       rigid_data.force,
                                                                       rigid_data.torque,
                                                                       rigid_data.body_indices,
                                                                       n_group_bodies,
                                                                       n_bodies,
                                                                       npt_rdata.epsilon_dot,
                                                                       npt_rdata.partial_Ksum_t,
                                                                       npt_rdata.partial_Ksum_r,
                                                                       npt_rdata.nf_t,
                                                                       npt_rdata.nf_r,
                                                                       npt_rdata.dimension,
                                                                       box,
                                                                       deltaT);

    gpu_nph_rigid_remap_kernel<<< body_grid, body_threads >>>(rigid_data.com,
                                                              rigid_data.body_indices,
                                                              n_group_bodies,
                                                              n_bodies,
                                                              box,
                                                              npt_rdata.dilation,
                                                              npt_rdata.dimension,
                                                              npt_rdata.new_box);


    return hipSuccess;
    }

#pragma mark RIGID_STEP_TWO_KERNEL
//! Takes the 2nd 1/2 step forward in the velocity-verlet NPH integration scheme
/*!
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total number of rigid bodies
    \param npt_rdata_eta_dot_t0 Thermostat translational part
    \param npt_rdata_eta_dot_r0 Thermostat rotational part
    \param npt_rdata_epsilon_dot Barostat velocity
    \param npt_rdata_partial_Ksum_t Body translational kinetic energy
    \param npt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param npt_rdata_nf_t Translational degrees of freedom
    \param npt_rdata_nf_r Translational degrees of freedom
    \param npt_rdata_dimension System dimesion
    \param deltaT Timestep
    \param box Box dimensions for periodic boundary condition handling
*/

extern "C" __global__ void gpu_nph_rigid_step_two_body_kernel(Scalar4* rdata_vel,
                                                              Scalar4* rdata_angmom,
                                                              Scalar4* rdata_angvel,
                                                              Scalar4* rdata_orientation,
                                                              Scalar4* rdata_conjqm,
                                                              Scalar *d_rigid_mass,
                                                              Scalar4 *d_rigid_mi,
                                                              Scalar4 *d_rigid_force,
                                                              Scalar4 *d_rigid_torque,
                                                              unsigned int *d_rigid_group,
                                                              unsigned int n_group_bodies,
                                                              unsigned int n_bodies,
                                                              Scalar npt_rdata_epsilon_dot,
                                                              Scalar* npt_rdata_partial_Ksum_t,
                                                              Scalar* npt_rdata_partial_Ksum_r,
                                                              unsigned int npt_rdata_nf_t,
                                                              unsigned int npt_rdata_nf_r,
                                                              unsigned int npt_rdata_dimension,
                                                              BoxDim box,
                                                              Scalar deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= n_group_bodies)
        return;

    Scalar body_mass;
    Scalar4 moment_inertia, vel, ex_space, ey_space, ez_space, orientation, conjqm;
    Scalar4 force, torque;
    Scalar4 mbody, tbody, fquat;

    Scalar dt_half = Scalar(0.5) * deltaT;
    Scalar onednft, onednfr, tmp, scale_t, scale_r, akin_t, akin_r;

    onednft = Scalar(1.0) + (Scalar) (npt_rdata_dimension) / (Scalar) (npt_rdata_nf_t+npt_rdata_nf_r);
    onednfr = (Scalar) (npt_rdata_dimension) / (Scalar) (npt_rdata_nf_t+npt_rdata_nf_r);

    tmp = Scalar(-1.0) * dt_half * onednft * npt_rdata_epsilon_dot;
    scale_t = exp(tmp);
    tmp = Scalar(-1.0) * dt_half * onednfr * npt_rdata_epsilon_dot;
    scale_r = exp(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];

    // Update body velocity and angmom
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    vel = rdata_vel[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    orientation = rdata_orientation[idx_body];
    conjqm = rdata_conjqm[idx_body];

    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);

    Scalar dtfm = dt_half / body_mass;

    // update the velocity
    Scalar4 vel2;
    vel2.x = scale_t * vel.x + dtfm * force.x;
    vel2.y = scale_t * vel.y + dtfm * force.y;
    vel2.z = scale_t * vel.z + dtfm * force.z;
    vel2.w = 0;

    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;

    // update angular momentum
    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);

    Scalar4  conjqm2, angmom2;
    conjqm2.x = scale_r * conjqm.x + deltaT * fquat.x;
    conjqm2.y = scale_r * conjqm.y + deltaT * fquat.y;
    conjqm2.z = scale_r * conjqm.z + deltaT * fquat.z;
    conjqm2.w = scale_r * conjqm.w + deltaT * fquat.w;

    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);

    angmom2.x *= Scalar(0.5);
    angmom2.y *= Scalar(0.5);
    angmom2.z *= Scalar(0.5);
    angmom2.w = 0;

    // update angular velocity
    Scalar4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);

    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;

    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_conjqm[idx_body] = conjqm2;

    npt_rdata_partial_Ksum_t[group_idx] = akin_t;
    npt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param d_net_virial Particle net virial
    \param box Box dimensions for periodic boundary condition handling
    \param npt_rdata Thermostat/barostat data
    \param deltaT Amount of real time to step forward in one time step

*/
hipError_t gpu_nph_rigid_step_two( const gpu_rigid_data_arrays& rigid_data,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    Scalar4 *d_net_force,
                                    Scalar *d_net_virial,
                                    const BoxDim& box,
                                    const gpu_npt_rigid_data& npt_rdata,
                                    Scalar deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;

    unsigned int block_size = 64;
    unsigned int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);
    gpu_nph_rigid_step_two_body_kernel<<< body_grid, body_threads >>>(rigid_data.vel,
                                                                    rigid_data.angmom,
                                                                    rigid_data.angvel,
                                                                    rigid_data.orientation,
                                                                    rigid_data.conjqm,
                                                                    rigid_data.body_mass,
                                                                    rigid_data.moment_inertia,
                                                                    rigid_data.force,
                                                                    rigid_data.torque,
                                                                    rigid_data.body_indices,
                                                                    n_group_bodies,
                                                                    n_bodies,
                                                                    npt_rdata.epsilon_dot,
                                                                    npt_rdata.partial_Ksum_t,
                                                                    npt_rdata.partial_Ksum_r,
                                                                    npt_rdata.nf_t,
                                                                    npt_rdata.nf_r,
                                                                    npt_rdata.dimension,
                                                                    box,
                                                                    deltaT);


    return hipSuccess;
    }

#pragma mark RIGID_KINETIC_ENERGY_REDUCTION

//! Shared memory for kinetic energy reduction
extern __shared__ Scalar nph_rigid_sdata[];

/*! Summing the kinetic energy of rigid bodies
    \param npt_rdata Thermostat data for rigid bodies

*/
extern "C" __global__ void gpu_nph_rigid_reduce_ksum_kernel(Scalar* npt_rdata_partial_Ksum_t,
                                                            Scalar* npt_rdata_partial_Ksum_r,
                                                            Scalar* npt_rdata_Ksum_t,
                                                            Scalar* npt_rdata_Ksum_r,
                                                            unsigned int n_bodies)
    {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar* body_ke_t = nph_rigid_sdata;
    Scalar* body_ke_r = &nph_rigid_sdata[blockDim.x];

    Scalar Ksum_t = 0, Ksum_r=0;

    // sum up the values in the partial sum via a sliding window
    for (int start = 0; start < n_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < n_bodies)
            {
            body_ke_t[threadIdx.x] = npt_rdata_partial_Ksum_t[start + threadIdx.x];
            body_ke_r[threadIdx.x] = npt_rdata_partial_Ksum_r[start + threadIdx.x];
            }
        else
            {
            body_ke_t[threadIdx.x] = 0;
            body_ke_r[threadIdx.x] = 0;
            }
        __syncthreads();

        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_ke_t[threadIdx.x] += body_ke_t[threadIdx.x + offset];
                body_ke_r[threadIdx.x] += body_ke_r[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }

        // everybody sums up Ksum
        Ksum_t += body_ke_t[0];
        Ksum_r += body_ke_r[0];
        }

    __syncthreads();


    if (global_idx == 0)
        {
        *npt_rdata_Ksum_t = Ksum_t;
        *npt_rdata_Ksum_r = Ksum_r;
        }

    }

/*!
    \param npt_rdata Thermostat/barostat data for rigid bodies
*/
hipError_t gpu_nph_rigid_reduce_ksum(const gpu_npt_rigid_data& npt_rdata)
    {
    // setup the grid to run the kernel
    int block_size = 128;
    dim3 grid( 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel: double the block size to accomodate Ksum_t and Ksum_r
    gpu_nph_rigid_reduce_ksum_kernel<<< grid, threads, 2 * block_size * sizeof(Scalar) >>>(npt_rdata.partial_Ksum_t,
                                                                                          npt_rdata.partial_Ksum_r,
                                                                                          npt_rdata.Ksum_t,
                                                                                          npt_rdata.Ksum_r,
                                                                                          npt_rdata.n_bodies);

    return hipSuccess;
    }
