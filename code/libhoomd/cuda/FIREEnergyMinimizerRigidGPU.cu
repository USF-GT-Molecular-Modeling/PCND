#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: ndtrung

#include "FIREEnergyMinimizerRigidGPU.cuh"
#include "TextureTools.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

#include <stdio.h>

/*! \file FIREEnergyMinimizerRigidGPU.cu
    \brief Defines GPU kernel code for one performing one FIRE energy
    minimization iteration on the GPU. Used by FIREEnergyMinimizerRigidGPU.
*/

//! The texture for reading the rigid data body indices array
texture<unsigned int, 1, hipReadModeElementType> rigid_data_body_indices_tex;

//! The texture for reading the rigid data vel array
scalar4_tex_t rigid_data_vel_tex;
//! The texture for reading the rigid data angvel array
scalar4_tex_t rigid_data_angvel_tex;
//! The texture for reading the rigid data angmom array
scalar4_tex_t rigid_data_angmom_tex;
//! The texture for reading the rigid data force array
scalar4_tex_t rigid_data_force_tex;
//! The texture for reading the rigid data torque array
scalar4_tex_t rigid_data_torque_tex;
//! The texture for reading the net force array
scalar4_tex_t net_force_tex;

//! Shared memory used in reducing sums
extern __shared__ Scalar fire_sdata[];

#pragma mark ZERO_VELOCITY_KERNEL

//! The kernel function to zeros velocities, called by gpu_fire_rigid_zero_v()
/*! \param rdata_vel Body velocities
    \param rdata_angmom Angular momenta
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Number of rigid bodies
*/
extern "C" __global__ void gpu_fire_rigid_zero_v_kernel(Scalar4* rdata_vel,
                                                Scalar4* rdata_angmom,
                                                unsigned int n_group_bodies,
                                                unsigned int n_bodies)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < n_group_bodies)
        {
        unsigned int idx_body = tex1Dfetch(rigid_data_body_indices_tex, group_idx);
        Scalar4 vel = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
        Scalar4 angmom = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

        if (idx_body < n_bodies)
            {
            rdata_vel[idx_body] = vel;
            rdata_angmom[idx_body] = angmom;
            }
        }
    }


/*! \param rdata Rigid data to zero velocities for

This function is just the driver for gpu_fire_rigid_zero_v_kernel(), see that function
for details.
*/
hipError_t gpu_fire_rigid_zero_v(gpu_rigid_data_arrays rdata)
    {
    unsigned int n_group_bodies = rdata.n_group_bodies;
    unsigned int n_bodies = rdata.n_bodies;

    // setup the grid to run the kernel
    unsigned int block_size = 256;
    unsigned int num_blocks = n_group_bodies / block_size + 1;
    dim3 grid(num_blocks, 1, 1);
    dim3 threads(block_size, 1, 1);

    hipError_t error = hipBindTexture(0, rigid_data_body_indices_tex, rdata.body_indices, sizeof(Scalar) * n_group_bodies);
    if (error != hipSuccess)
        return error;

    // run the kernel
    gpu_fire_rigid_zero_v_kernel<<< grid, threads >>>(rdata.vel, rdata.angmom, n_group_bodies, n_bodies);

    return hipSuccess;
    }

#pragma mark SUMMING_POWER_KERNEL

/*! Kernel function to simultaneously compute the partial sum over Pt, vsq and fsq for the FIRE algorithm
    \param d_sum_Pt Array to hold the sum over Pt (f*v), v2 and f2
    \param rdata_force The developer has chosen not to document this parameter
    \param rdata_vel The developer has chosen not to document this parameter
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Number of rigid bodies
*/
extern "C" __global__ void gpu_fire_rigid_reduce_Pt_kernel(Scalar* d_sum_Pt,
                                                            Scalar4* rdata_force,
                                                            Scalar4* rdata_vel,
                                                            unsigned int n_group_bodies,
                                                            unsigned int n_bodies)
    {
    unsigned int idx_global = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar* body_Pt = fire_sdata;
    Scalar* body_vsq = &fire_sdata[blockDim.x];
    Scalar* body_fsq = &fire_sdata[2*blockDim.x];

    Scalar4 force, vel;
    Scalar Pt = Scalar(0.0);
    Scalar vsq = Scalar(0.0);
    Scalar fsq = Scalar(0.0);

    // sum up the values via a sliding window
    for (int start = 0; start < n_group_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < n_group_bodies)
            {
            unsigned int idx_body = tex1Dfetch(rigid_data_body_indices_tex, start + threadIdx.x);
            Scalar Ptrans = Scalar(0.0);
            Scalar v2 = Scalar(0.0);
            Scalar f2 = 0.0;

            if (idx_body < n_bodies)
                {
                force = texFetchScalar4(rdata_force, rigid_data_force_tex, idx_body);
                vel = texFetchScalar4(rdata_vel, rigid_data_vel_tex, idx_body);
                Ptrans = force.x * vel.x + force.y * vel.y + force.z * vel.z;
                v2 = vel.x * vel.x + vel.y * vel.y + vel.z * vel.z;
                f2 = force.x * force.x + force.y * force.y + force.z * force.z;
                }

            body_Pt[threadIdx.x] = Ptrans;
            body_vsq[threadIdx.x] = v2;
            body_fsq[threadIdx.x] = f2;
            }
        else
            {
            body_Pt[threadIdx.x] = Scalar(0.0);
            body_vsq[threadIdx.x] = Scalar(0.0);
            body_fsq[threadIdx.x] = Scalar(0.0);
            }

        __syncthreads();

        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_Pt[threadIdx.x] += body_Pt[threadIdx.x + offset];
                body_vsq[threadIdx.x] += body_vsq[threadIdx.x + offset];
                body_fsq[threadIdx.x] += body_fsq[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }

        // everybody sums up to the local variables
        Pt += body_Pt[0];
        vsq += body_vsq[0];
        fsq += body_fsq[0];
        }

    __syncthreads();

    // only one thread write to the global memory
    if (idx_global == 0)
        {
        d_sum_Pt[0] = Pt;
        d_sum_Pt[1] = vsq;
        d_sum_Pt[2] = fsq;
        }
    }

/*! Kernel function to simultaneously compute the partial sum over Pr, wsq and tsq for the FIRE algorithm
    \param d_sum_Pr Array to hold the sum over Pr (t*w), w2 and t2
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Number of rigid bodies
    \param rdata_torque The developer has chosen not to document this variable
    \param rdata_angvel The developer has chosen not to document this variable
*/
extern "C" __global__ void gpu_fire_rigid_reduce_Pr_kernel(Scalar* d_sum_Pr,
                                                            Scalar4* rdata_torque,
                                                            Scalar4* rdata_angvel,
                                                            unsigned int n_group_bodies,
                                                            unsigned int n_bodies)
    {
    unsigned int idx_global = blockDim.x * blockIdx.x + threadIdx.x;

    Scalar* body_Pr = fire_sdata;
    Scalar* body_wsq = &fire_sdata[blockDim.x];
    Scalar* body_tsq = &fire_sdata[2*blockDim.x];

    Scalar4 torque, angvel;
    Scalar Pr = Scalar(0.0);
    Scalar wsq = Scalar(0.0);
    Scalar tsq = Scalar(0.0);

    // sum up the values via a sliding window
    for (unsigned int start = 0; start < n_group_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < n_group_bodies)
            {
            unsigned int idx_body = tex1Dfetch(rigid_data_body_indices_tex, start + threadIdx.x);
            Scalar Prot = Scalar(0.0);
            Scalar w2 = Scalar(0.0);
            Scalar t2 = Scalar(0.0);

            if (idx_body < n_bodies)
                {
                torque = texFetchScalar4(rdata_torque, rigid_data_torque_tex, idx_body);
                angvel = texFetchScalar4(rdata_angvel, rigid_data_angvel_tex, idx_body);
                Prot = torque.x * angvel.x + torque.y * angvel.y + torque.z * angvel.z;
                w2 = angvel.x * angvel.x + angvel.y * angvel.y + angvel.z * angvel.z;
                t2 = torque.x * torque.x + torque.y * torque.y + torque.z * torque.z;
                }

            body_Pr[threadIdx.x] = Prot;
            body_wsq[threadIdx.x] = w2;
            body_tsq[threadIdx.x] = t2;
            }
        else
            {
            body_Pr[threadIdx.x] = Scalar(0.0);
            body_wsq[threadIdx.x] = Scalar(0.0);
            body_tsq[threadIdx.x] = Scalar(0.0);
            }

        __syncthreads();

        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_Pr[threadIdx.x] += body_Pr[threadIdx.x + offset];
                body_wsq[threadIdx.x] += body_wsq[threadIdx.x + offset];
                body_tsq[threadIdx.x] += body_tsq[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }

        // everybody sums up to the local variables
        Pr += body_Pr[0];
        wsq += body_wsq[0];
        tsq += body_tsq[0];
        }

    __syncthreads();

    // only one thread write to the global memory
    if (idx_global == 0)
        {
        d_sum_Pr[0] = Pr;
        d_sum_Pr[1] = wsq;
        d_sum_Pr[2] = tsq;
        }
    }


/*! Summing the translational and rotational powers across the rigid bodies
    \param rdata Rigid data to compute the sums for
    \param d_sum_Pt Array to hold the sum over Pt
    \param d_sum_Pr Array to hold the sum over Pr
*/
hipError_t gpu_fire_rigid_compute_sum_all(const gpu_rigid_data_arrays& rdata,
                                        Scalar* d_sum_Pt,
                                        Scalar* d_sum_Pr)
    {
    unsigned int n_bodies = rdata.n_bodies;
    unsigned int n_group_bodies = rdata.n_group_bodies;

    hipError_t error = hipBindTexture(0, rigid_data_body_indices_tex, rdata.body_indices, sizeof(Scalar) * n_group_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_vel_tex, rdata.vel, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_angvel_tex, rdata.angvel, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_force_tex, rdata.force, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_torque_tex, rdata.torque, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    // setup the grid to run the kernel
    unsigned int block_size = 128;
    dim3 grid( 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernels
    gpu_fire_rigid_reduce_Pt_kernel<<< grid, threads, 3 * block_size * sizeof(Scalar) >>>(d_sum_Pt, rdata.force, rdata.vel, n_group_bodies, n_bodies);

    gpu_fire_rigid_reduce_Pr_kernel<<< grid, threads, 3 * block_size * sizeof(Scalar) >>>(d_sum_Pr, rdata.torque, rdata.angvel, n_group_bodies, n_bodies);


    return hipSuccess;
    }


#pragma mark UPDATE_VELOCITY_KERNEL

//! Kernel function to update the velocties used by the FIRE algorithm
/*! \param rdata_vel Body velocities to be updated
    \param rdata_angmom Angular momenta to be updated
    \param rdata_force The developer has chosen not to document this variable
    \param rdata_torque The developer has chosen not to document this variable
    \param alpha Alpha coupling parameter used by the FIRE algorithm
    \param factor_t factor equal to alpha*vnorm/fnorm
    \param factor_r factor equal to alpha*wnorm/tnorm
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Number of rigid bodies
    \param local_beg Starting body index in this card
*/
extern "C" __global__ void gpu_fire_rigid_update_v_kernel(Scalar4* rdata_vel,
                                                        Scalar4* rdata_angmom,
                                                        Scalar4* rdata_force,
                                                        Scalar4* rdata_torque,
                                                        Scalar alpha,
                                                        Scalar factor_t,
                                                        Scalar factor_r,
                                                        unsigned int n_group_bodies,
                                                        unsigned int n_bodies,
                                                        unsigned int local_beg)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x + local_beg;

    if (group_idx < n_group_bodies)
        {
        unsigned int idx_body = tex1Dfetch(rigid_data_body_indices_tex, group_idx);
        if (idx_body < n_bodies)
            {
            // read the body data (MEM TRANSFER: 32 bytes)
            Scalar4 vel = texFetchScalar4(rdata_vel, rigid_data_vel_tex, idx_body);
            Scalar4 angmom = texFetchScalar4(rdata_angmom, rigid_data_angmom_tex, idx_body);
            Scalar4 force = texFetchScalar4(rdata_force, rigid_data_force_tex, idx_body);
            Scalar4 torque = texFetchScalar4(rdata_torque, rigid_data_torque_tex, idx_body);

            Scalar4 vel2;
            vel2.x = vel.x * (Scalar(1.0) - alpha) + force.x * factor_t;
            vel2.y = vel.y * (Scalar(1.0) - alpha) + force.y * factor_t;
            vel2.z = vel.z * (Scalar(1.0) - alpha) + force.z * factor_t;

            Scalar4 angmom2;
            angmom2.x = angmom.x * (Scalar(1.0) - alpha) + torque.x * factor_r;
            angmom2.y = angmom.y * (Scalar(1.0) - alpha) + torque.y * factor_r;
            angmom2.z = angmom.z * (Scalar(1.0) - alpha) + torque.z * factor_r;

            // write out the results (MEM_TRANSFER: 32 bytes)
            rdata_vel[idx_body] = vel2;
            rdata_angmom[idx_body] = angmom2;
            }
        }
    }

/*! \param rdata Rigid data to update the velocities for
    \param alpha Alpha coupling parameter used by the FIRE algorithm
    \param factor_t factor equal to alpha*vnorm/fnorm
    \param factor_r factor equal to alpha*wnorm/tnorm
    This function is a driver for gpu_fire_rigid_update_v_kernel(), see it for details.
*/
hipError_t gpu_fire_rigid_update_v(gpu_rigid_data_arrays rdata,
                                                    Scalar alpha,
                                                    Scalar factor_t,
                                                    Scalar factor_r)
    {
    unsigned int n_bodies = rdata.n_bodies;
    unsigned int n_group_bodies = rdata.n_group_bodies;
    unsigned int local_beg = rdata.local_beg;

    // setup the grid to run the kernel
    unsigned int block_size = 256;
    unsigned int num_blocks = n_group_bodies / block_size + 1;
    dim3 grid(num_blocks, 1, 1);
    dim3 threads(block_size, 1, 1);

    hipError_t error = hipBindTexture(0, rigid_data_body_indices_tex, rdata.body_indices, sizeof(Scalar) * n_group_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_vel_tex, rdata.vel, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_angmom_tex, rdata.angmom, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_force_tex, rdata.force, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
        return error;

    error = hipBindTexture(0, rigid_data_torque_tex, rdata.torque, sizeof(Scalar4) * n_bodies);
    if (error != hipSuccess)
    return error;

    // run the kernel
    gpu_fire_rigid_update_v_kernel<<< grid, threads >>>(rdata.vel,
                                                    rdata.angmom,
                                                    rdata.force,
                                                    rdata.torque,
                                                    alpha,
                                                    factor_t,
                                                    factor_r,
                                                    n_group_bodies,
                                                    n_bodies,
                                                    local_beg);

    return hipSuccess;
    }
