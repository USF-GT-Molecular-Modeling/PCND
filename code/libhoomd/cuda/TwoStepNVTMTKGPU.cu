#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

#include "TwoStepNVTMTKGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNVTGPU.cu
    \brief Defines GPU kernel code for NVT integration on the GPU. Used by TwoStepNVTGPU.
*/

//! Takes the first 1/2 step forward in the NVT integration step
/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param exp_fac Velocity rescaling factor from thermostat
    \param deltaT Amount of real time to step forward in one time step

    Take the first half step forward in the NVT integration.

    See gpu_nve_step_one_kernel() for some performance notes on how to handle the group data reads efficiently.
*/
extern "C" __global__
void gpu_nvt_mtk_step_one_kernel(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             BoxDim box,
                             Scalar exp_fac,
                             Scalar deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // update positions to the next timestep and update velocities to the next half step
        Scalar4 postype = d_pos[idx];
        Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);

        Scalar4 velmass = d_vel[idx];
        Scalar3 vel = make_scalar3(velmass.x, velmass.y, velmass.z);
        Scalar3 accel = d_accel[idx];

        // perform update computation
        vel = vel*exp_fac + Scalar(1.0/2.0) * accel * deltaT;
        pos += vel * deltaT;

        // read in the image flags
        int3 image = d_image[idx];

        // time to fix the periodic boundary conditions
        box.wrap(pos, image);

        // write out the results
        d_pos[idx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
        d_vel[idx] = make_scalar4(vel.x, vel.y, vel.z, velmass.w);
        d_image[idx] = image;
        }
    }

/*! \param d_pos array of particle positions
    \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_image array of particle images
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param box Box dimensions for periodic boundary condition handling
    \param block_size Size of the block to run
    \param exp_fac Thermostat rescaling factor
    \param deltaT Amount of real time to step forward in one time step
*/
hipError_t gpu_nvt_mtk_step_one(Scalar4 *d_pos,
                             Scalar4 *d_vel,
                             const Scalar3 *d_accel,
                             int3 *d_image,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             const BoxDim& box,
                             unsigned int block_size,
                             Scalar exp_fac,
                             Scalar deltaT)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nvt_mtk_step_one_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid( (group_size/run_block_size) + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);

    // run the kernel
    gpu_nvt_mtk_step_one_kernel<<< grid, threads >>>(d_pos,
                         d_vel,
                         d_accel,
                         d_image,
                         d_group_members,
                         group_size,
                         box,
                         exp_fac,
                         deltaT);
    return hipSuccess;
    }

//! Takes the second 1/2 step forward in the NVT integration step
/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param deltaT Amount of real time to step forward in one time step
*/
extern "C" __global__
void gpu_nvt_mtk_step_two_kernel(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar4 *d_net_force,
                             Scalar deltaT)
    {
    // determine which particle this thread works on
    int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx < group_size)
        {
        unsigned int idx = d_group_members[group_idx];

        // read in the net force and calculate the acceleration
        Scalar4 net_force = d_net_force[idx];
        Scalar3 accel = make_scalar3(net_force.x,net_force.y,net_force.z);

        Scalar4 vel = d_vel[idx];
        Scalar3 v = make_scalar3(vel.x,vel.y,vel.z);

        Scalar mass = vel.w;
        accel = accel/mass;
        v += Scalar(1.0/2.0) * deltaT * accel;

        // write out data
        d_vel[idx] = make_scalar4(v.x,v.y,v.z,vel.w);

        // since we calculate the acceleration, we need to write it for the next step
        d_accel[idx] = accel;
        }
    }

/*! \param d_vel array of particle velocities
    \param d_accel array of particle accelerations
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Net force on each particle
    \param block_size Size of the block to execute on the device
    \param deltaT Amount of real time to step forward in one time step
*/
hipError_t gpu_nvt_mtk_step_two(Scalar4 *d_vel,
                             Scalar3 *d_accel,
                             unsigned int *d_group_members,
                             unsigned int group_size,
                             Scalar4 *d_net_force,
                             unsigned int block_size,
                             Scalar deltaT)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nvt_mtk_step_two_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid( (group_size/run_block_size) + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);

    // run the kernel
    gpu_nvt_mtk_step_two_kernel<<< grid, threads >>>(d_vel, d_accel, d_group_members, group_size, d_net_force, deltaT);

    return hipSuccess;
    }

// vim:syntax=cpp
