#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


// Maintainer: ndtrung

#include "QuaternionMath.h"
#include "TwoStepNVTRigidGPU.cuh"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

/*! \file TwoStepNVTRigidGPU.cu
    \brief Defines GPU kernel code for NVT integration on the GPU. Used by TwoStepNVTRigidGPU.
*/

// Flag for invalid particle index, identical to the sentinel value NO_INDEX in RigidData.h
#define INVALID_INDEX 0xffffffff

/*! Taylor expansion
    \param x Point to take the expansion

*/
__device__ Scalar taylor_exp(Scalar x)
    {
    Scalar x2, x3, x4, x5;
    x2 = x * x;
    x3 = x2 * x;
    x4 = x2 * x2;
    x5 = x4 * x;
    return (Scalar(1.0) + x + x2 / Scalar(2.0) + x3 / Scalar(6.0) + x4 / Scalar(24.0) + x5 / Scalar(120.0));
    }

#pragma mark RIGID_STEP_ONE_KERNEL
/*! Takes the first half-step forward for rigid bodies in the velocity-verlet NVT integration
    \param rdata_com Body center of mass
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_body_image Body image
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param n_group_bodies Number of rigid bodies in my group
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total umber of rigid bodies
    \param nvt_rdata_eta_dot_t0 Thermostat translational part
    \param nvt_rdata_eta_dot_r0 Thermostat rotational part
    \param nvt_rdata_partial_Ksum_t Body translational kinetic energy
    \param nvt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param deltaT Timestep
    \param box Box dimensions for periodic boundary condition handling
*/

extern "C" __global__ void gpu_nvt_rigid_step_one_body_kernel(Scalar4* rdata_com,
                                                            Scalar4* rdata_vel,
                                                            Scalar4* rdata_angmom,
                                                            Scalar4* rdata_angvel,
                                                            Scalar4* rdata_orientation,
                                                            int3* rdata_body_image,
                                                            Scalar4* rdata_conjqm,
                                                            Scalar *d_rigid_mass,
                                                            Scalar4 *d_rigid_mi,
                                                            Scalar4 *d_rigid_force,
                                                            Scalar4 *d_rigid_torque,
                                                            unsigned int *d_rigid_group,
                                                            unsigned int n_group_bodies,
                                                            unsigned int n_bodies,
                                                            Scalar nvt_rdata_eta_dot_t0,
                                                            Scalar nvt_rdata_eta_dot_r0,
                                                            Scalar* nvt_rdata_partial_Ksum_t,
                                                            Scalar* nvt_rdata_partial_Ksum_r,
                                                            BoxDim box,
                                                            Scalar deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // do velocity verlet update
    // v(t+deltaT/2) = v(t) + (1/2)a*deltaT
    // r(t+deltaT) = r(t) + v(t+deltaT/2)*deltaT
    if (group_idx >= n_group_bodies)
        return;

    Scalar body_mass;
    Scalar4 moment_inertia, com, vel, orientation, ex_space, ey_space, ez_space, force, torque, conjqm;
    int3 body_image;
    Scalar4 mbody, tbody, fquat;

    Scalar dt_half = Scalar(0.5) * deltaT;
    Scalar   tmp, scale_t, scale_r, akin_t, akin_r;
    tmp = -Scalar(1.0) * dt_half * nvt_rdata_eta_dot_t0;
    scale_t = fast::exp(tmp);
    tmp = -Scalar(1.0) * dt_half * nvt_rdata_eta_dot_r0;
    scale_r = fast::exp(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    com = rdata_com[idx_body];
    vel = rdata_vel[idx_body];
    orientation = rdata_orientation[idx_body];
    body_image = rdata_body_image[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    conjqm = rdata_conjqm[idx_body];

    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);

    // update velocity
    Scalar dtfm = dt_half / body_mass;

    Scalar4 vel2;
    vel2.x = vel.x + dtfm * force.x;
    vel2.y = vel.y + dtfm * force.y;
    vel2.z = vel.z + dtfm * force.z;
    vel2.x *= scale_t;
    vel2.y *= scale_t;
    vel2.z *= scale_t;
    vel2.w = vel.w;

    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;

    // update position
    Scalar3 pos2;
    pos2.x = com.x + vel2.x * deltaT;
    pos2.y = com.y + vel2.y * deltaT;
    pos2.z = com.z + vel2.z * deltaT;

    // time to fix the periodic boundary conditions
    box.wrap(pos2, body_image);

    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);

    Scalar4 conjqm2;
    conjqm2.x = conjqm.x + deltaT * fquat.x;
    conjqm2.y = conjqm.y + deltaT * fquat.y;
    conjqm2.z = conjqm.z + deltaT * fquat.z;
    conjqm2.w = conjqm.w + deltaT * fquat.w;

    conjqm2.x *= scale_r;
    conjqm2.y *= scale_r;
    conjqm2.z *= scale_r;
    conjqm2.w *= scale_r;

    // step 1.4 to 1.13 - use no_squish rotate to update p and q
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(1, conjqm2, orientation, moment_inertia, deltaT);
    no_squish_rotate(2, conjqm2, orientation, moment_inertia, dt_half);
    no_squish_rotate(3, conjqm2, orientation, moment_inertia, dt_half);

    // update the exyz_space
    // transform p back to angmom
    // update angular velocity
    Scalar4 angmom2;
    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);
    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);

    angmom2.x *= Scalar(0.5);
    angmom2.y *= Scalar(0.5);
    angmom2.z *= Scalar(0.5);

    Scalar4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);

    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;

    // write out the results (MEM_TRANSFER: ? bytes)
    rdata_com[idx_body] = make_scalar4(pos2.x, pos2.y, pos2.z, com.w);
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_orientation[idx_body] = orientation;
    rdata_body_image[idx_body] = body_image;
    rdata_conjqm[idx_body] = conjqm2;

    nvt_rdata_partial_Ksum_t[group_idx] = akin_t;
    nvt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param box Box dimensions for periodic boundary condition handling
    \param nvt_rdata Thermostat data
    \param deltaT Amount of real time to step forward in one time step

*/
hipError_t gpu_nvt_rigid_step_one( const gpu_rigid_data_arrays& rigid_data,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    Scalar4 *d_net_force,
                                    const BoxDim& box,
                                    const gpu_nvt_rigid_data& nvt_rdata,
                                    Scalar deltaT)
    {
    assert(d_net_force);

    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;

    // setup the grid to run the kernel for rigid bodies
    int block_size = 64;
    int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);

    gpu_nvt_rigid_step_one_body_kernel<<< body_grid, body_threads  >>>(rigid_data.com,
                                                            rigid_data.vel,
                                                            rigid_data.angmom,
                                                            rigid_data.angvel,
                                                            rigid_data.orientation,
                                                            rigid_data.body_image,
                                                            rigid_data.conjqm,
                                                            rigid_data.body_mass,
                                                            rigid_data.moment_inertia,
                                                            rigid_data.force,
                                                            rigid_data.torque,
                                                            rigid_data.body_indices,
                                                            n_group_bodies,
                                                            n_bodies,
                                                            nvt_rdata.eta_dot_t0,
                                                            nvt_rdata.eta_dot_r0,
                                                            nvt_rdata.partial_Ksum_t,
                                                            nvt_rdata.partial_Ksum_r,
                                                            box,
                                                            deltaT);


    return hipSuccess;
    }

#pragma mark RIGID_STEP_TWO_KERNEL


//! Takes the 2nd 1/2 step forward in the velocity-verlet NVT integration scheme
/*!
    \param rdata_vel Body velocity
    \param rdata_angmom Angular momentum
    \param rdata_angvel Angular velocity
    \param rdata_orientation Quaternion
    \param rdata_conjqm Conjugate quaternion momentum
    \param d_rigid_mass Body mass
    \param d_rigid_mi Body inertia moments
    \param d_rigid_force Body forces
    \param d_rigid_torque Body torques
    \param d_rigid_group Body indices
    \param n_group_bodies Number of rigid bodies in my group
    \param n_bodies Total number of rigid bodies
    \param nvt_rdata_eta_dot_t0 Thermostat translational part
    \param nvt_rdata_eta_dot_r0 Thermostat rotational part
    \param nvt_rdata_partial_Ksum_t Body translational kinetic energy
    \param nvt_rdata_partial_Ksum_r Body rotation kinetic energy
    \param deltaT Timestep
    \param box Box dimensions for periodic boundary condition handling
*/

extern "C" __global__ void gpu_nvt_rigid_step_two_body_kernel(Scalar4* rdata_vel,
                                                          Scalar4* rdata_angmom,
                                                          Scalar4* rdata_angvel,
                                                          Scalar4* rdata_orientation,
                                                          Scalar4* rdata_conjqm,
                                                          Scalar *d_rigid_mass,
                                                          Scalar4 *d_rigid_mi,
                                                          Scalar4 *d_rigid_force,
                                                          Scalar4 *d_rigid_torque,
                                                          unsigned int *d_rigid_group,
                                                          unsigned int n_group_bodies,
                                                          unsigned int n_bodies,
                                                          Scalar nvt_rdata_eta_dot_t0,
                                                          Scalar nvt_rdata_eta_dot_r0,
                                                          Scalar* nvt_rdata_partial_Ksum_t,
                                                          Scalar* nvt_rdata_partial_Ksum_r,
                                                          BoxDim box,
                                                          Scalar deltaT)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (group_idx >= n_group_bodies)
        return;

    Scalar body_mass;
    Scalar4 moment_inertia, vel, ex_space, ey_space, ez_space, orientation, conjqm;
    Scalar4 force, torque;
    Scalar4 mbody, tbody, fquat;

    Scalar dt_half = Scalar(0.5) * deltaT;
    Scalar   tmp, scale_t, scale_r, akin_t, akin_r;
    tmp = -Scalar(1.0) * dt_half * nvt_rdata_eta_dot_t0;
    scale_t = fast::exp(tmp);
    tmp = -Scalar(1.0) * dt_half * nvt_rdata_eta_dot_r0;
    scale_r = fast::exp(tmp);

    unsigned int idx_body = d_rigid_group[group_idx];

    // Update body velocity and angmom
    body_mass = d_rigid_mass[idx_body];
    moment_inertia = d_rigid_mi[idx_body];
    vel = rdata_vel[idx_body];
    force = d_rigid_force[idx_body];
    torque = d_rigid_torque[idx_body];
    orientation = rdata_orientation[idx_body];
    conjqm = rdata_conjqm[idx_body];

    exyzFromQuaternion(orientation, ex_space, ey_space, ez_space);

    Scalar dtfm = dt_half / body_mass;

    // update the velocity
    Scalar4 vel2;
    vel2.x = scale_t * vel.x + dtfm * force.x;
    vel2.y = scale_t * vel.y + dtfm * force.y;
    vel2.z = scale_t * vel.z + dtfm * force.z;
    vel2.w = Scalar(0.0);

    tmp = vel2.x * vel2.x + vel2.y * vel2.y + vel2.z * vel2.z;
    akin_t = body_mass * tmp;

    // update angular momentum
    matrix_dot(ex_space, ey_space, ez_space, torque, tbody);
    quatvec(orientation, tbody, fquat);

    Scalar4  conjqm2, angmom2;
    conjqm2.x = scale_r * conjqm.x + deltaT * fquat.x;
    conjqm2.y = scale_r * conjqm.y + deltaT * fquat.y;
    conjqm2.z = scale_r * conjqm.z + deltaT * fquat.z;
    conjqm2.w = scale_r * conjqm.w + deltaT * fquat.w;

    invquatvec(orientation, conjqm2, mbody);
    transpose_dot(ex_space, ey_space, ez_space, mbody, angmom2);

    angmom2.x *= Scalar(0.5);
    angmom2.y *= Scalar(0.5);
    angmom2.z *= Scalar(0.5);
    angmom2.w = Scalar(0.0);

    // update angular velocity
    Scalar4 angvel2;
    computeAngularVelocity(angmom2, moment_inertia, ex_space, ey_space, ez_space, angvel2);

    akin_r = angmom2.x * angvel2.x + angmom2.y * angvel2.y + angmom2.z * angvel2.z;

    // write out results
    rdata_vel[idx_body] = vel2;
    rdata_angmom[idx_body] = angmom2;
    rdata_angvel[idx_body] = angvel2;
    rdata_conjqm[idx_body] = conjqm2;

    nvt_rdata_partial_Ksum_t[group_idx] = akin_t;
    nvt_rdata_partial_Ksum_r[group_idx] = akin_r;
    }

/*! \param rigid_data Rigid body data to step forward 1/2 step
    \param d_group_members Device array listing the indicies of the mebers of the group to integrate
    \param group_size Number of members in the group
    \param d_net_force Particle net forces
    \param d_net_virial Particle net virial
    \param box Box dimensions for periodic boundary condition handling
    \param nvt_rdata Thermostat data
    \param deltaT Amount of real time to step forward in one time step

*/
hipError_t gpu_nvt_rigid_step_two( const gpu_rigid_data_arrays& rigid_data,
                                    unsigned int *d_group_members,
                                    unsigned int group_size,
                                    Scalar4 *d_net_force,
                                    Scalar *d_net_virial,
                                    const BoxDim& box,
                                    const gpu_nvt_rigid_data& nvt_rdata,
                                    Scalar deltaT)
    {
    unsigned int n_bodies = rigid_data.n_bodies;
    unsigned int n_group_bodies = rigid_data.n_group_bodies;

    unsigned int block_size = 64;
    unsigned int n_blocks = n_group_bodies / block_size + 1;
    dim3 body_grid(n_blocks, 1, 1);
    dim3 body_threads(block_size, 1, 1);
    gpu_nvt_rigid_step_two_body_kernel<<< body_grid, body_threads >>>(rigid_data.vel,
                                                                rigid_data.angmom,
                                                                rigid_data.angvel,
                                                                rigid_data.orientation,
                                                                rigid_data.conjqm,
                                                                rigid_data.body_mass,
                                                                rigid_data.moment_inertia,
                                                                rigid_data.force,
                                                                rigid_data.torque,
                                                                rigid_data.body_indices,
                                                                n_group_bodies,
                                                                n_bodies,
                                                                nvt_rdata.eta_dot_t0,
                                                                nvt_rdata.eta_dot_r0,
                                                                nvt_rdata.partial_Ksum_t,
                                                                nvt_rdata.partial_Ksum_r,
                                                                box,
                                                                deltaT);


    return hipSuccess;
    }

#pragma mark RIGID_KINETIC_ENERGY_REDUCTION

//! Shared memory for kinetic energy reduction
extern __shared__ Scalar nvt_rigid_sdata[];

/*! Summing the kinetic energy of rigid bodies
    \param nvt_rdata Thermostat data for rigid bodies

*/
extern "C" __global__ void gpu_nvt_rigid_reduce_ksum_kernel(gpu_nvt_rigid_data nvt_rdata)
    {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    Scalar* body_ke_t = nvt_rigid_sdata;
    Scalar* body_ke_r = &nvt_rigid_sdata[blockDim.x];

    Scalar Ksum_t = Scalar(0.0), Ksum_r=Scalar(0.0);

    // sum up the values in the partial sum via a sliding window
    for (int start = 0; start < nvt_rdata.n_bodies; start += blockDim.x)
        {
        if (start + threadIdx.x < nvt_rdata.n_bodies)
            {
            body_ke_t[threadIdx.x] = nvt_rdata.partial_Ksum_t[start + threadIdx.x];
            body_ke_r[threadIdx.x] = nvt_rdata.partial_Ksum_r[start + threadIdx.x];
            }
        else
            {
            body_ke_t[threadIdx.x] = Scalar(0.0);
            body_ke_r[threadIdx.x] = Scalar(0.0);
            }
        __syncthreads();

        // reduce the sum within a block
        int offset = blockDim.x >> 1;
        while (offset > 0)
            {
            if (threadIdx.x < offset)
                {
                body_ke_t[threadIdx.x] += body_ke_t[threadIdx.x + offset];
                body_ke_r[threadIdx.x] += body_ke_r[threadIdx.x + offset];
                }
            offset >>= 1;
            __syncthreads();
            }

        // everybody sums up Ksum
        Ksum_t += body_ke_t[0];
        Ksum_r += body_ke_r[0];
        }

    __syncthreads();


    if (global_idx == 0)
        {
        *nvt_rdata.Ksum_t = Ksum_t;
        *nvt_rdata.Ksum_r = Ksum_r;
        }

    }

/*!
    \param nvt_rdata Thermostat data for rigid bodies

*/
hipError_t gpu_nvt_rigid_reduce_ksum(const gpu_nvt_rigid_data& nvt_rdata)
    {
    // setup the grid to run the kernel
    int block_size = 128;
    dim3 grid( 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel: double the block size to accomodate Ksum_t and Ksum_r
    gpu_nvt_rigid_reduce_ksum_kernel<<< grid, threads, 2 * block_size * sizeof(Scalar) >>>(nvt_rdata);

    return hipSuccess;
    }
