#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: joaander

#include "NeighborListGPUBinned.cuh"
#include "TextureTools.h"

/*! \file NeighborListGPUBinned.cu
    \brief Defines GPU kernel code for O(N) neighbor list generation on the GPU
*/

//! Texture for reading d_cell_xyzf
scalar4_tex_t cell_xyzf_1d_tex;

//! Warp-centric scan
template<int NT>
struct warp_scan
    {
    #if __CUDA_ARCH__ >= 300
    enum { capacity = 0 }; // uses no shared memory
    #else
    enum { capacity = NT > 1 ? (2 * NT + 1) : 1};
    #endif

    __device__ static int Scan(int tid, unsigned char x, volatile unsigned char *shared, unsigned char* total)
        {
        #if __CUDA_ARCH__ >= 300
        // Kepler version
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));

        int first = laneid - tid;

        #pragma unroll
        for(int offset = 1; offset < NT; offset += offset)
            {
            int y = __shfl(x,(first + tid - offset) &(WARP_SIZE -1));
            if(tid >= offset) x += y;
            }

        // all threads get the total from the last thread in the cta
        *total = __shfl(x,first + NT - 1);

        // shift by one (exclusive scan)
        int y = __shfl(x,(first + tid - 1) &(WARP_SIZE-1));
        x = tid ? y : 0;

        #else // __CUDA_ARCH__ >= 300

        shared[tid] = x;
        int first = 0;
        // no syncthreads here (inside warp)

        for(int offset = 1; offset < NT; offset += offset)
            {
            if(tid >= offset)
                x = shared[first + tid - offset] + x;
            first = NT - first;
            shared[first + tid] = x;
            // no syncthreads here (inside warp)
            }
        *total = shared[first + NT - 1];

        // shift by one (exclusive scan)
        x = tid ? shared[first + tid - 1] : 0;
        #endif
        // no syncthreads here (inside warp)
        return x;
        }
    };

//! Kernel call for generating neighbor list on the GPU (shared memory version)
/*! \tparam flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering.
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param nli Indexer to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param d_cell_size Number of particles in each cell
    \param d_cell_xyzf Cell contents (xyzf array from CellList with flag=type)
    \param d_cell_tdb Cell contents (tdb array from CellList with)
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer for indexing cells
    \param cli Cell list indexer for indexing into d_cell_xyzf
    \param cadji Adjacent cell indexer listing the 27 neighboring cells
    \param box Simulation box dimensions
    \param r_maxsq The maximum radius for which to include particles as neighbors, squared
    \param r_max The maximum radius for which to include particles as neighbors
    \param ghost_width Width of ghost cell layer

    \note optimized for Fermi
*/
template<unsigned char flags, int threads_per_particle>
__global__ void gpu_compute_nlist_binned_shared_kernel(unsigned int *d_nlist,
                                                    unsigned int *d_n_neigh,
                                                    Scalar4 *d_last_updated_pos,
                                                    unsigned int *d_conditions,
                                                    const Index2D nli,
                                                    const Scalar4 *d_pos,
                                                    const unsigned int *d_body,
                                                    const Scalar *d_diameter,
                                                    const unsigned int N,
                                                    const unsigned int *d_cell_size,
                                                    const Scalar4 *d_cell_xyzf,
                                                    const Scalar4 *d_cell_tdb,
                                                    const unsigned int *d_cell_adj,
                                                    const Index3D ci,
                                                    const Index2D cli,
                                                    const Index2D cadji,
                                                    const BoxDim box,
                                                    const Scalar r_maxsq,
                                                    const Scalar r_max,
                                                    const Scalar3 ghost_width)
    {
    bool filter_body = flags & 1;
    bool filter_diameter = flags & 2;

    // each set of threads_per_particle threads is going to compute the neighbor list for a single particle
    int my_pidx;
    if (gridDim.y > 1)
        {
        // fermi workaround
        my_pidx = (blockIdx.x + blockIdx.y*65535) * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }
    else
        {
        my_pidx = blockIdx.x * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }

    // return early if out of bounds
    if (my_pidx >= N) return;

    // first, determine which bin this particle belongs to
    Scalar4 my_postype = d_pos[my_pidx];
    Scalar3 my_pos = make_scalar3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_body = d_body[my_pidx];
    Scalar my_diameter = d_diameter[my_pidx];

    Scalar3 f = box.makeFraction(my_pos, ghost_width);

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // shared memory (volatile is required, since we are doing warp-centric)
    volatile extern __shared__ unsigned char sh[];

    // index of current neighbor
    unsigned int cur_adj = 0;

    // current cell
    unsigned int neigh_cell = d_cell_adj[cadji(cur_adj, my_cell)];

    // size of current cell
    unsigned int neigh_size = d_cell_size[neigh_cell];

    // offset of cta in shared memory
    int cta_offs = (threadIdx.x/threads_per_particle)*warp_scan<threads_per_particle>::capacity;

    // current index in cell
    int cur_offset = threadIdx.x % threads_per_particle;

    bool done = false;

    // total number of neighbors
    unsigned int nneigh = 0;

    while (! done)
        {
        // initalize with default
        unsigned int neighbor;
        unsigned char has_neighbor = 0;

        // advance neighbor cell
        while (cur_offset >= neigh_size && !done )
            {
            cur_offset -= neigh_size;
            cur_adj++;
            if (cur_adj < cadji.getW())
                {
                neigh_cell = d_cell_adj[cadji(cur_adj, my_cell)];
                neigh_size = d_cell_size[neigh_cell];
                }
            else
                // we are past the end of the cell neighbors
                done = true;
            }

        // if the first thread in the cta has no work, terminate the loop
        if (done && !(threadIdx.x % threads_per_particle)) break;

        if (!done)
            {
            Scalar4 cur_xyzf = texFetchScalar4(d_cell_xyzf, cell_xyzf_1d_tex, cli(cur_offset, neigh_cell));

            Scalar4 cur_tdb = make_scalar4(0, 0, 0, 0);
            if (filter_diameter || filter_body)
                cur_tdb = d_cell_tdb[cli(cur_offset, neigh_cell)];

            // advance cur_offset
            cur_offset += threads_per_particle;

            unsigned int neigh_body = __scalar_as_int(cur_tdb.z);
            Scalar neigh_diameter = cur_tdb.y;

            Scalar3 neigh_pos = make_scalar3(cur_xyzf.x,
                                           cur_xyzf.y,
                                           cur_xyzf.z);
            int cur_neigh = __scalar_as_int(cur_xyzf.w);

            // compute the distance between the two particles
            Scalar3 dx = my_pos - neigh_pos;

            // wrap the periodic boundary conditions
            dx = box.minImage(dx);

            // compute dr squared
            Scalar drsq = dot(dx,dx);

            bool excluded = (my_pidx == cur_neigh);

            if (filter_body && my_body != 0xffffffff)
                excluded = excluded | (my_body == neigh_body);

            Scalar sqshift(0.0);
            if (filter_diameter)
                {
                // compute the shift in radius to accept neighbors based on their diameters
                Scalar delta = (my_diameter + neigh_diameter) * Scalar(0.5) - Scalar(1.0);
                // r^2 < (r_max + delta)^2
                // r^2 < r_maxsq + delta^2 + 2*r_max*delta
                sqshift = (delta + Scalar(2.0) * r_max) * delta;
                }

            // store result in shared memory
            if (drsq <= (r_maxsq + sqshift) && !excluded)
                {
                neighbor = cur_neigh;
                has_neighbor = 1;
                }
            }

        // no syncthreads here, we assume threads_per_particle < warp size

        // scan over flags
        unsigned char n;
        int k = warp_scan<threads_per_particle>::Scan(threadIdx.x % threads_per_particle,
            has_neighbor, &sh[cta_offs], &n);

        if (has_neighbor && nneigh + k < nli.getH())
            d_nlist[nli(my_pidx, nneigh + k)] = neighbor;

        // increment total neighbor count
        nneigh += n;
        } // end while

    if (threadIdx.x % threads_per_particle == 0)
        {
        // flag if we need to grow the neighbor list
        if (nneigh >= nli.getH())
            atomicMax(&d_conditions[0], nneigh);

        d_n_neigh[my_pidx] = nneigh;
        d_last_updated_pos[my_pidx] = my_postype;
        }
    }

//! determine maximum possible block size
template<typename T>
int get_max_block_size(T func)
    {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))func);
    int max_threads = attr.maxThreadsPerBlock;
    // number of threads has to be multiple of warp size
    max_threads -= max_threads % max_threads_per_particle;
    return max_threads;
    }

void gpu_nlist_binned_bind_texture(const Scalar4 *d_cell_xyzf, unsigned int n_elements)
    {
    // bind the position texture
    cell_xyzf_1d_tex.normalized = false;
    cell_xyzf_1d_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, cell_xyzf_1d_tex, d_cell_xyzf, sizeof(Scalar4)*n_elements);
    }

//! recursive template to launch neighborlist with given template parameters
/* \tparam cur_tpp Number of threads per particle (assumed to be power of two) */
template<int cur_tpp>
inline void launcher(unsigned int *d_nlist,
              unsigned int *d_n_neigh,
              Scalar4 *d_last_updated_pos,
              unsigned int *d_conditions,
              const Index2D nli,
              const Scalar4 *d_pos,
              const unsigned int *d_body,
              const Scalar *d_diameter,
              const unsigned int N,
              const unsigned int *d_cell_size,
              const Scalar4 *d_cell_xyzf,
              const Scalar4 *d_cell_tdb,
              const unsigned int *d_cell_adj,
              const Index3D ci,
              const Index2D cli,
              const Index2D cadji,
              const BoxDim box,
              const Scalar r_maxsq,
              const Scalar r_max,
              const Scalar3 ghost_width,
              const unsigned int compute_capability,
              unsigned int tpp,
              bool filter_diameter,
              bool filter_body,
              unsigned int block_size)
    {
    unsigned int shared_size = 0;

    if (tpp == cur_tpp && cur_tpp != 0)
        {
        if (!filter_diameter && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_shared_kernel<0,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            if (compute_capability < 30) shared_size = warp_scan<cur_tpp>::capacity*sizeof(unsigned char)*(block_size/cur_tpp);

            gpu_compute_nlist_binned_shared_kernel<0,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                             d_n_neigh,
                                                                             d_last_updated_pos,
                                                                             d_conditions,
                                                                             nli,
                                                                             d_pos,
                                                                             d_body,
                                                                             d_diameter,
                                                                             N,
                                                                             d_cell_size,
                                                                             d_cell_xyzf,
                                                                             d_cell_tdb,
                                                                             d_cell_adj,
                                                                             ci,
                                                                             cli,
                                                                             cadji,
                                                                             box,
                                                                             r_maxsq,
                                                                             sqrtf(r_maxsq),
                                                                             ghost_width);
            }
        else if (!filter_diameter && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_shared_kernel<1,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            if (compute_capability < 30) shared_size = warp_scan<cur_tpp>::capacity*sizeof(unsigned char)*(block_size/cur_tpp);

            gpu_compute_nlist_binned_shared_kernel<1,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                             d_n_neigh,
                                                                             d_last_updated_pos,
                                                                             d_conditions,
                                                                             nli,
                                                                             d_pos,
                                                                             d_body,
                                                                             d_diameter,
                                                                             N,
                                                                             d_cell_size,
                                                                             d_cell_xyzf,
                                                                             d_cell_tdb,
                                                                             d_cell_adj,
                                                                             ci,
                                                                             cli,
                                                                             cadji,
                                                                             box,
                                                                             r_maxsq,
                                                                             sqrtf(r_maxsq),
                                                                             ghost_width);
            }
        else if (filter_diameter && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_shared_kernel<2,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            if (compute_capability < 30) shared_size = warp_scan<cur_tpp>::capacity*sizeof(unsigned char)*(block_size/cur_tpp);

            gpu_compute_nlist_binned_shared_kernel<2,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                             d_n_neigh,
                                                                             d_last_updated_pos,
                                                                             d_conditions,
                                                                             nli,
                                                                             d_pos,
                                                                             d_body,
                                                                             d_diameter,
                                                                             N,
                                                                             d_cell_size,
                                                                             d_cell_xyzf,
                                                                             d_cell_tdb,
                                                                             d_cell_adj,
                                                                             ci,
                                                                             cli,
                                                                             cadji,
                                                                             box,
                                                                             r_maxsq,
                                                                             sqrtf(r_maxsq),
                                                                             ghost_width);
            }
        else if (filter_diameter && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_shared_kernel<3,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            if (compute_capability < 30) shared_size = warp_scan<cur_tpp>::capacity*sizeof(unsigned char)*(block_size/cur_tpp);

            gpu_compute_nlist_binned_shared_kernel<3,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                             d_n_neigh,
                                                                             d_last_updated_pos,
                                                                             d_conditions,
                                                                             nli,
                                                                             d_pos,
                                                                             d_body,
                                                                             d_diameter,
                                                                             N,
                                                                             d_cell_size,
                                                                             d_cell_xyzf,
                                                                             d_cell_tdb,
                                                                             d_cell_adj,
                                                                             ci,
                                                                             cli,
                                                                             cadji,
                                                                             box,
                                                                             r_maxsq,
                                                                             sqrtf(r_maxsq),
                                                                             ghost_width);
            }
        }
    else
        {
        launcher<cur_tpp/2>(d_nlist,
                     d_n_neigh,
                     d_last_updated_pos,
                     d_conditions,
                     nli,
                     d_pos,
                     d_body,
                     d_diameter,
                     N,
                     d_cell_size,
                     d_cell_xyzf,
                     d_cell_tdb,
                     d_cell_adj,
                     ci,
                     cli,
                     cadji,
                     box,
                     r_maxsq,
                     sqrtf(r_maxsq),
                     ghost_width,
                     compute_capability,
                     tpp,
                     filter_diameter,
                     filter_body,
                     block_size
                     );
        }
    }

//! template specialization to terminate recursion
template<>
inline void launcher<min_threads_per_particle/2>(unsigned int *d_nlist,
              unsigned int *d_n_neigh,
              Scalar4 *d_last_updated_pos,
              unsigned int *d_conditions,
              const Index2D nli,
              const Scalar4 *d_pos,
              const unsigned int *d_body,
              const Scalar *d_diameter,
              const unsigned int N,
              const unsigned int *d_cell_size,
              const Scalar4 *d_cell_xyzf,
              const Scalar4 *d_cell_tdb,
              const unsigned int *d_cell_adj,
              const Index3D ci,
              const Index2D cli,
              const Index2D cadji,
              const BoxDim box,
              const Scalar r_maxsq,
              const Scalar r_max,
              const Scalar3 ghost_width,
              const unsigned int compute_capability,
              unsigned int tpp,
              bool filter_diameter,
              bool filter_body,
              unsigned int block_size)
    { }

hipError_t gpu_compute_nlist_binned_shared(unsigned int *d_nlist,
                                     unsigned int *d_n_neigh,
                                     Scalar4 *d_last_updated_pos,
                                     unsigned int *d_conditions,
                                     const Index2D& nli,
                                     const Scalar4 *d_pos,
                                     const unsigned int *d_body,
                                     const Scalar *d_diameter,
                                     const unsigned int N,
                                     const unsigned int *d_cell_size,
                                     const Scalar4 *d_cell_xyzf,
                                     const Scalar4 *d_cell_tdb,
                                     const unsigned int *d_cell_adj,
                                     const Index3D& ci,
                                     const Index2D& cli,
                                     const Index2D& cadji,
                                     const BoxDim& box,
                                     const Scalar r_maxsq,
                                     const unsigned int threads_per_particle,
                                     const unsigned int block_size,
                                     bool filter_body,
                                     bool filter_diameter,
                                     const Scalar3& ghost_width,
                                     const unsigned int compute_capability)
    {
    launcher<max_threads_per_particle>(d_nlist,
                                   d_n_neigh,
                                   d_last_updated_pos,
                                   d_conditions,
                                   nli,
                                   d_pos,
                                   d_body,
                                   d_diameter,
                                   N,
                                   d_cell_size,
                                   d_cell_xyzf,
                                   d_cell_tdb,
                                   d_cell_adj,
                                   ci,
                                   cli,
                                   cadji,
                                   box,
                                   r_maxsq,
                                   sqrtf(r_maxsq),
                                   ghost_width,
                                   compute_capability,
                                   threads_per_particle,
                                   filter_diameter,
                                   filter_body,
                                   block_size
                                   );

    return hipSuccess;
    }

// don't compile the 1x nlist kernel in double precision builds
#ifdef SINGLE_PRECISION
//! Texture for reading d_cell_adj
texture<unsigned int, 2, hipReadModeElementType> cell_adj_tex;
//! Texture for reading d_cell_size
texture<unsigned int, 1, hipReadModeElementType> cell_size_tex;
//! Texture for reading d_cell_xyzf
texture<Scalar4, 2, hipReadModeElementType> cell_xyzf_tex;
//! Texture for reading d_cell_tdb
texture<Scalar4, 2, hipReadModeElementType> cell_tdb_tex;

//! Kernel call for generating neighbor list on the GPU
/*! \tparam filter_flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering.
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param nli Indexer to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param ci Cell indexer for indexing cells
    \param box Simulation box dimensions
    \param r_maxsq The maximum radius for which to include particles as neighbors, squared
    \param r_max The maximum radius for which to include particles as neighbors
    \param ghost_width Width of ghost cell layer

    \note optimized for compute 1.x devices
*/
template<unsigned char filter_flags>
__global__ void gpu_compute_nlist_binned_1x_kernel(unsigned int *d_nlist,
                                                   unsigned int *d_n_neigh,
                                                   Scalar4 *d_last_updated_pos,
                                                   unsigned int *d_conditions,
                                                   const Index2D nli,
                                                   const Scalar4 *d_pos,
                                                   const unsigned int *d_body,
                                                   const Scalar *d_diameter,
                                                   const unsigned int N,
                                                   const Index3D ci,
                                                   const BoxDim box,
                                                   const float r_maxsq,
                                                   const float r_max,
                                                   const Scalar3 ghost_width)
    {
    bool filter_body = filter_flags & 1;
    bool filter_diameter = filter_flags & 2;

    // each thread is going to compute the neighbor list for a single particle
    int my_pidx = blockDim.x * blockIdx.x + threadIdx.x;

    // count the number of neighbors needed
    unsigned int n_neigh_needed = 0;

    // quit early if we are past the end of the array
    if (my_pidx >= N)
        return;

    // first, determine which bin this particle belongs to
    Scalar4 my_postype = d_pos[my_pidx];
    Scalar3 my_pos = make_scalar3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_body = d_body[my_pidx];
    Scalar my_diameter = d_diameter[my_pidx];

    // get periodic flags
    uchar3 periodic = box.getPeriodic();

    // find the bin each particle belongs in
    Scalar3 f = box.makeFraction(my_pos,ghost_width);
    unsigned int ib = (unsigned int)(f.x * ci.getW());
    unsigned int jb = (unsigned int)(f.y * ci.getH());
    unsigned int kb = (unsigned int)(f.z * ci.getD());

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // each thread will determine the neighborlist of a single particle
    // count number of neighbors found so far in n_neigh
    int n_neigh = 0;

    // loop over all adjacent bins
    for (unsigned int cur_adj = 0; cur_adj < 27; cur_adj++)
        {
        int neigh_cell = tex2D(cell_adj_tex, cur_adj, my_cell);
        unsigned int size = tex1Dfetch(cell_size_tex, neigh_cell);

        Scalar4 next_xyzf = tex2D(cell_xyzf_tex, 0, neigh_cell);

        // now, we are set to loop through the array
        for (int cur_offset = 0; cur_offset < size; cur_offset++)
            {
            Scalar4 cur_xyzf = next_xyzf;
            next_xyzf = tex2D(cell_xyzf_tex, cur_offset+1, neigh_cell);
            Scalar4 cur_tdb = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
            if (filter_diameter || filter_body)
                cur_tdb = tex2D(cell_tdb_tex, cur_offset, neigh_cell);
            unsigned int neigh_body = __scalar_as_int(cur_tdb.z);
            Scalar neigh_diameter = cur_tdb.y;

            Scalar3 neigh_pos = make_scalar3(cur_xyzf.x,
                                           cur_xyzf.y,
                                           cur_xyzf.z);
            int cur_neigh = __scalar_as_int(cur_xyzf.w);

            // compute the distance between the two particles
            Scalar3 dx = my_pos - neigh_pos;

            // wrap the periodic boundary conditions
            dx = box.minImage(dx);
            // compute dr squared
            Scalar drsq = dot(dx,dx);


            bool excluded = (my_pidx == cur_neigh);

            if (filter_body && my_body != 0xffffffff)
                excluded = excluded | (my_body == neigh_body);

            Scalar sqshift = Scalar(0.0);
            if (filter_diameter)
                {
                // compute the shift in radius to accept neighbors based on their diameters
                Scalar delta = (my_diameter + neigh_diameter) * Scalar(0.5) - Scalar(1.0);
                // r^2 < (r_max + delta)^2
                // r^2 < r_maxsq + delta^2 + 2*r_max*delta
                sqshift = (delta + Scalar(2.0) * r_max) * delta;
                }

            if (drsq <= (r_maxsq + sqshift) && !excluded)
                {
                if (n_neigh < nli.getH())
                    d_nlist[nli(my_pidx, n_neigh)] = cur_neigh;
                else
                    n_neigh_needed = n_neigh+1;

                n_neigh++;
                }
            }
        }

    d_n_neigh[my_pidx] = n_neigh;
    d_last_updated_pos[my_pidx] = my_postype;

    if (n_neigh_needed > 0)
        atomicMax(&d_conditions[0], n_neigh_needed);
    }
#endif  // #ifdef SINGLE_PRECISION

hipError_t gpu_compute_nlist_binned_1x(unsigned int *d_nlist,
                                        unsigned int *d_n_neigh,
                                        Scalar4 *d_last_updated_pos,
                                        unsigned int *d_conditions,
                                        const Index2D& nli,
                                        const Scalar4 *d_pos,
                                        const unsigned int *d_body,
                                        const Scalar *d_diameter,
                                        const unsigned int N,
                                        const unsigned int *d_cell_size,
                                        const hipArray *dca_cell_xyzf,
                                        const hipArray *dca_cell_tdb,
                                        const hipArray *dca_cell_adj,
                                        const Index3D& ci,
                                        const BoxDim& box,
                                        const Scalar r_maxsq,
                                        const unsigned int block_size,
                                        bool filter_body,
                                        bool filter_diameter,
                                        const Scalar3& ghost_width)
    {
    // don't compile the 1x nlist kernel in double precision builds
    #ifdef SINGLE_PRECISION
    int n_blocks = (int)ceil(double(N)/double(block_size));

    hipError_t err = hipBindTextureToArray(cell_adj_tex, dca_cell_adj);
    if (err != hipSuccess)
        return err;

    err = hipBindTextureToArray(cell_xyzf_tex, dca_cell_xyzf);
    if (err != hipSuccess)
        return err;

    err = hipBindTextureToArray(cell_tdb_tex, dca_cell_tdb);
    if (err != hipSuccess)
        return err;

    err = hipBindTexture(0, cell_size_tex, d_cell_size, sizeof(unsigned int)*ci.getNumElements());
    if (err != hipSuccess)
        return err;

    if (!filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<0><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (!filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<1><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (filter_diameter && !filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<2><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width);
        }
    if (filter_diameter && filter_body)
        {
        gpu_compute_nlist_binned_1x_kernel<3><<<n_blocks, block_size>>>(d_nlist,
                                                                        d_n_neigh,
                                                                        d_last_updated_pos,
                                                                        d_conditions,
                                                                        nli,
                                                                        d_pos,
                                                                        d_body,
                                                                        d_diameter,
                                                                        N,
                                                                        ci,
                                                                        box,
                                                                        r_maxsq,
                                                                        sqrtf(r_maxsq),
                                                                        ghost_width );
        }
    #endif // #ifdef SINGLE_PRECISION

    return hipSuccess;
    }
